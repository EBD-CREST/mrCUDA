#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_SAFE_CALL(x) \
{ \
    if ((x) != hipSuccess) { \
        fprintf(stderr, "Error!");   \
        exit(EXIT_FAILURE); \
    } \
}

int main()
{
    float *a;
    CUDA_SAFE_CALL(hipMalloc(&a, sizeof(float)));
    printf("a is %p\n", a);
    getchar();
    return 0;
}

