#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_SAFE_CALL(x) \
{ \
    if ((x) != hipSuccess) { \
        fprintf(stderr, "Error!");   \
        exit(EXIT_FAILURE); \
    } \
}

int main()
{
    float *a, *b;
    CUDA_SAFE_CALL(hipSetDevice(0));
    CUDA_SAFE_CALL(hipMalloc(&a, sizeof(float)));
    CUDA_SAFE_CALL(hipSetDevice(1));
    CUDA_SAFE_CALL(hipMalloc(&b, sizeof(float)));
    printf("a on device 0 is %p\n", a);
    printf("b on device 1 is %p\n", b);
    return 0;
}

